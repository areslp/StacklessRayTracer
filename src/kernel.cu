#include "hip/hip_runtime.h"
//kernelPBO.cu (Rob Farber)

#include <stdio.h>
#include "scene.hpp"
#include "kdtree.h"

#define blocksize_x 8
#define blocksize_y 6 

#define EPSILON (1e-6)

__constant__ float3 dcamPos;
__constant__ float3 dwincenterPos;

__constant__ float3 dwinup;
__constant__ float3 dwindown;
__constant__ float3 dboxup;
__constant__ float3 dboxdown;

__constant__ unsigned int dwinwidth;
__constant__ unsigned int dwinheight;

__constant__ float hx;
__constant__ float hy;

__constant__ float3 objectColor; 

__constant__ unsigned int nTri;
__constant__ unsigned int nVtx;
__constant__ unsigned int nLS;

/*__constant__ float* dVtxBuf;
__constant__ int* dTriVtxBuf;
__constant__ float* dNormal;
__constant__ float* dLS;
__constant__ float* dsandboxColor;
__constant__ unsigned int* dsandboxIsReflective;
*/

texture<float, 1 > texref_VtxBuf;
texture<int, 1 > texref_TriVtx;
texture<float, 1 > texref_Normal;
texture<float, 1 > texref_LS;
texture<float, 1 > texref_sandboxColor;
texture<unsigned int, 1 > texref_sandboxIsReflective;

// memory address in cudamemory
/*
extern "C" float* dVtxBuf = NULL;
extern "C" int* dTriVtxBuf = NULL;
extern "C" float* dNormal = NULL;
extern "C" float* dLS = NULL;
extern "C" float* dsandboxColor = NULL;
extern "C" unsigned int* dsandboxIsReflective = NULL;
*/

void checkCUDAError(const char *msg) 
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

__device__ inline float3 CrossProduct( float3 p1, float3 p2)
{
  float3 tmp;
  tmp.x = p1.y*p2.z - p1.z*p2.y;
  tmp.y = p1.z*p2.x - p1.x*p2.z;
//  tmp.y = p1.x*p2.z - p1.z*p2.x;
  tmp.z = p1.x*p2.y - p1.y*p2.x;
  return tmp;
}

__device__ inline  float InnerProduct( float3 p1, float3 p2)
{
  return p1.x*p2.x + p1.y*p2.y + p1.z*p2.z;
}

// return the value of t, when p+t*dir intersect triangle p0-p1-p2
__device__ inline float TestSingleIntersection( float3 p, float3 dir, float3 p0, float3 p1, float3 p2) 
{
  float3 E1;
  E1.x = p1.x - p0.x;
  E1.y = p1.y - p0.y;
  E1.z = p1.z - p0.z;

  float3 E2;
  E2.x = p2.x - p0.x;
  E2.y = p2.y - p0.y;
  E2.z = p2.z - p0.z;

  float3 T;
  T.x = p.x - p0.x;
  T.y = p.y - p0.y;
  T.z = p.z - p0.z;

  float3 P = CrossProduct( dir, E2);
  float3 Q = CrossProduct( T, E1 );
  
  float s = InnerProduct(P, E1);
 // s = 0.5; //FIXME
  if ( (s < EPSILON ) && (s>-EPSILON) ) return -1;

  float t = InnerProduct(Q, E2) / s;
  if ( t < 1e-5) return -2;

  float u = InnerProduct(P, T) / s;
  if ( (u < 0)  ) return -3;

  float v = InnerProduct(Q, dir) / s;
  if ( (v < 0 ) ) return -4;

  if ( u+v > 1 ) return -5;

//  t = 0.5; //FIXME
  return t;
}

__device__ inline bool FromLeftToRight( KdTree_rp* dtree, int idx, float3 dir)
{
  switch (dtree[idx].axis)
  {
    case 0:
      if ( dir.x > EPSILON) return true;
      else return false;
    case 1:
      if ( dir.y > EPSILON) return true;
      else return false;
    case 2:
      if ( dir.z > EPSILON) return true;
      else return false;
    default:
      printf("no such case!\n");
  }
}

__device__ inline bool TestNodeIntersection( KdTree_rp* dtree, int idx, float3 p, float3 dir)
{
  
  float tmp1, tmp2;

  // x-y
  if ( (dir.x<EPSILON) && (dir.x>-EPSILON) )
  {
    if ( (p.x>dtree[idx].xmax) || ( p.x<dtree[idx].xmin) ) return false;
  }
  else
  {
    float tx1 = (dtree[idx].xmin-p.x)/dir.x;
   // if ( tx1<EPSILON ) return false;
    tmp1 = p.y + tx1*dir.y;

    float tx2 = (dtree[idx].xmax-p.x)/dir.x;
   //if ( tx2 < EPSILON ) return false;
    tmp2 = p.y + tx2*dir.y;
  
    if ( ( (tmp1>dtree[idx].ymax) && (tmp2>dtree[idx].ymax))
      || ( (tmp1<dtree[idx].ymin) && (tmp2<dtree[idx].ymin) ) ) return false;
  }

  // y-z
  if ( (dir.y<EPSILON) && (dir.y>-EPSILON) )
  {
    if ( (p.y>dtree[idx].ymax) || (p.y<dtree[idx].ymin) ) return false;
  }
  else
  {
    float ty1 = (dtree[idx].ymin-p.y)/dir.y;
  //  if ( ty1 < EPSILON ) return false;
    tmp1 = p.z + ty1*dir.z;
  
    float ty2 = (dtree[idx].ymax-p.y)/dir.y;
   // if ( ty2 < EPSILON ) return false;
    tmp2 = p.z + ty2*dir.z;

    if ( ( (tmp1>dtree[idx].zmax) && (tmp2>dtree[idx].zmax))
      || ( (tmp1<dtree[idx].zmin) && (tmp2<dtree[idx].zmin) ) ) return false;
  }

  // x-z
  if ( (dir.z<EPSILON) && (dir.z>-EPSILON) )
  {
    if ( (p.z>dtree[idx].zmax) || (p.z<dtree[idx].zmin) ) return false;
  }
  else
  {
    float tz1 = (dtree[idx].zmin - p.z)/dir.z;
//    if ( tz1 < EPSILON ) return false;
    tmp1 = p.x + tz1*dir.x;

    float tz2 = (dtree[idx].zmax - p.z)/dir.z;
 //   if ( tz2 < EPSILON ) return false;
    tmp2 = p.x + tz2*dir.x;

    if ( ( (tmp1>dtree[idx].xmax) && (tmp2>dtree[idx].xmax))
      || ( (tmp1<dtree[idx].xmin) && (tmp2<dtree[idx].xmin) ) ) return false;
  }

  return true;
}

__device__ inline void IntersectBox( float3 p, float3 dir, KdTree_rp* dtree, int idx, float3& pt, int& face, int prevface)
{
  float t = 1e6;
  float tmp = 0;

  float ep = 0;
  float ep2 = 1e-6;

  face =  -1;

  float a,b;

  if ( (dir.x > 1e-5) || ( dir.x< -1e-5) )
  {
    if ( prevface != 0)
    {
      tmp = ( dtree[idx].xmin -  p.x) /dir.x; 
      a = p.y + tmp* dir.y;
      b = p.z + tmp* dir.z;
  
      if ( (tmp > -ep2) && (tmp<t) ) 
        if ( (a>=dtree[idx].ymin-ep) && (a<=dtree[idx].ymax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
        t =tmp;
        face = 0;
      }
    }

    if ( prevface != 1)
    {
      tmp = ( dtree[idx].xmax - p.x) /dir.x;
      a = p.y + tmp* dir.y;
      b = p.z + tmp* dir.z;
      if ( (tmp > -ep2) && (tmp<t) ) 
        if ( (a>=dtree[idx].ymin-ep) && (a<=dtree[idx].ymax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
        t =tmp;
        face = 1;
      }
    }
  }

  if ( (dir.y > 1e-5) || ( dir.y < -1e-5) )
  {
    if ( prevface != 2)
    {
      tmp = ( dtree[idx].ymin - p.y) / dir.y;
      a = p.x + tmp * dir.x;
      b = p.z + tmp * dir.z;
      if ( (tmp > -ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
          t =tmp;
          face = 2;
      }
    }

    if ( prevface != 3 )
    {
      tmp = ( dtree[idx].ymax - p.y) / dir.y;
      a = p.x + tmp * dir.x;
      b = p.z + tmp * dir.z;
      if ( (tmp > -ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
        t =tmp;
        face = 3;
      }
    }
  }

  if ( (dir.z > 1e-5) || ( dir.z < -1e-5) )
  {
    if ( prevface != 4 )
    {
      tmp = ( dtree[idx].zmin - p.z) / dir.z;
      a = p.x + tmp * dir.x;
      b = p.y + tmp * dir.y;
      if ( (tmp > -ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].ymin-ep) && (b<=dtree[idx].ymax+ep) ) 
      {
        t =tmp;
        face = 4;
      }
    }

    if ( prevface != 5)
    {
      tmp = ( dtree[idx].zmax - p.z) / dir.z;
      a = p.x + tmp * dir.x;
      b = p.y + tmp * dir.y;
      if ( (tmp > -ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].ymin-ep) && (b<=dtree[idx].ymax+ep) ) 
      {
        t =tmp;
        face = 5;
      }
    }
  }
  
  if (  face >=0 )
  {
    pt.x = p.x + t * dir.x;
    pt.y = p.y + t*dir.y;
    pt.z = p.z + t*dir.z;
  }
  
}

__device__ inline void IntersectBox2( float3 p, float3 dir, KdTree_rp* dtree, int idx, float3& pt, int& face)
{
  float t = 1e6;
  float tmp = 0;

  float ep = 0;
  float ep2 = 0;

  face =  -1;

  float a,b;

  if ( (dir.x > 1e-5) || ( dir.x< -1e-5) )
  {
      tmp = ( dtree[idx].xmin -  p.x) /dir.x; 
      a = p.y + tmp* dir.y;
      b = p.z + tmp* dir.z;
  
      if ( (tmp > ep2) && (tmp<t) ) 
        if ( (a>=dtree[idx].ymin-ep) && (a<=dtree[idx].ymax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
        t =tmp;
        face = 0;
      }

      tmp = ( dtree[idx].xmax - p.x) /dir.x;
      a = p.y + tmp* dir.y;
      b = p.z + tmp* dir.z;
      if ( (tmp > ep2) && (tmp<t) ) 
        if ( (a>=dtree[idx].ymin-ep) && (a<=dtree[idx].ymax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
        t =tmp;
        face = 1;
      }
  }

  if ( (dir.y > 1e-5) || ( dir.y < -1e-5) )
  {
      tmp = ( dtree[idx].ymin - p.y) / dir.y;
      a = p.x + tmp * dir.x;
      b = p.z + tmp * dir.z;
      if ( (tmp > ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
          t =tmp;
          face = 2;
      }

      tmp = ( dtree[idx].ymax - p.y) / dir.y;
      a = p.x + tmp * dir.x;
      b = p.z + tmp * dir.z;
      if ( (tmp > ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].zmin-ep) && (b<=dtree[idx].zmax+ep) ) 
      {
        t =tmp;
        face = 3;
      }
  }

  if ( (dir.z > 1e-5) || ( dir.z < -1e-5) )
  {
      tmp = ( dtree[idx].zmin - p.z) / dir.z;
      a = p.x + tmp * dir.x;
      b = p.y + tmp * dir.y;
      if ( (tmp > ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].ymin-ep) && (b<=dtree[idx].ymax+ep) ) 
      {
        t =tmp;
        face = 4;
      }

      tmp = ( dtree[idx].zmax - p.z) / dir.z;
      a = p.x + tmp * dir.x;
      b = p.y + tmp * dir.y;
      if ( (tmp > ep2) && ( tmp < t) )
        if ( (a>=dtree[idx].xmin-ep) && (a<=dtree[idx].xmax+ep) && (b>=dtree[idx].ymin-ep) && (b<=dtree[idx].ymax+ep) ) 
      {
        t =tmp;
        face = 5;
      }
  }
  
  if (  face >=0 )
  {
    pt.x = p.x + t * dir.x;
    pt.y = p.y + t*dir.y;
    pt.z = p.z + t*dir.z;
  }
  
}

__device__ inline int FindLeaf( KdTree_rp* dtree, int idx, float3 pt)
{
  while ( dtree[idx].trinum <0 )
  {
    switch (dtree[idx].axis)
    {
      case 0:
        if ( pt.x < dtree[idx].splitpos ) idx = dtree[idx].left;
        else idx = dtree[idx].right;
        break;
      case 1:
        if ( pt.y < dtree[idx].splitpos ) idx = dtree[idx].left;
        else idx = dtree[idx].right;
        break;
      case 2:
        if ( pt.z < dtree[idx].splitpos ) idx = dtree[idx].left;
        else idx = dtree[idx].right;
        break;
    }
  }

  return idx;

}

__device__ inline  void FindIntersectedTriangle( float3 p, float3 dir, int& intersectedTri, float3& intersectedPt, float* dVtxBuf, int* dTriVtxBuf, KdTree_rp* dtree)
{
  float tmin = 1e6;
  float t = tmin;
  intersectedTri = -1;
  float3 p0;
  float3 p1;
  float3 p2;
  float3 pt;
  int idx0;
 
//  int stack[2*TREEDEPTH];
  
  int idx = 0;
  int face = -1;
  if ( (p.x>dtree[0].xmax) || ( p.y > dtree[0].ymax) || (p.z > dtree[0].zmax) 
    || ( p.x<dtree[0].xmin) || (p.y<dtree[0].ymin) || (p.z < dtree[0].zmin) )
    IntersectBox(p, dir, dtree, idx, pt, face, -1); // p is out of dtree[0]
  else // p is in dtree[0], find the leaf node containing p 
  {
    /*
    while (dtree[idx].trinum < 0 )
    {
      switch (dtree[idx].axis)
      {
        case 0:
          if ( p.x < dtree[idx].splitpos) idx = dtree[idx].left;
          else idx = dtree[idx].right;
        case 1:
          if ( p.y < dtree[idx].splitpos) idx = dtree[idx].left;
          else idx = dtree[idx].right;
        case 2:
          if ( p.z < dtree[idx].splitpos) idx = dtree[idx].left;
          else idx = dtree[idx].right;
      }
    }

      for ( int i = 0; i < dtree[idx].trinum; i++ )
      {
       // printf("%d ", dtree[idx].tri[i]);

        idx0 =  dTriVtxBuf[3*dtree[idx].tri[i]];

        p0.x = dVtxBuf[3*idx0];
        p0.y = dVtxBuf[3*idx0+1];
        p0.z = dVtxBuf[3*idx0+2];

        idx0  = dTriVtxBuf[3*dtree[idx].tri[i]+1];
        p1.x = dVtxBuf[3*idx0];
        p1.y = dVtxBuf[3*idx0+1];
        p1.z = dVtxBuf[3*idx0+2];

        idx0 = dTriVtxBuf[3*dtree[idx].tri[i]+2];
        p2.x = dVtxBuf[3*idx0];
        p2.y = dVtxBuf[3*idx0+1];
        p2.z = dVtxBuf[3*idx0+2];

        t = TestSingleIntersection( p, dir ,p0, p1, p2); 
    
        if ( (t > 1e-5) && ( t < tmin ) )
        {
          intersectedTri = idx;
          intersectedPt.x = p.x + t*dir.x;
          intersectedPt.y = p.y + t*dir.y;
          intersectedPt.z = p.z + t*dir.z;
          tmin = t;
        }
        
      }

    if ( intersectedTri >= 0 ) return;

    IntersectBox2(p, dir, dtree, idx, pt, face );
    idx = dtree[idx].rope[face];
    if ( (face==1) || (face==3) || (face==5) ) face--;
    else face++;
    */

    face = -2;
    pt = p;
  }
      
  if ( (face >= 0) || (face==-2) )
  while ( (intersectedTri<0) && (idx>=0) ) 
  {
    idx =  FindLeaf( dtree, idx, pt ); 

//    printf("%d\n",idx);
    if (dtree[idx].trinum>0) // leaf node
    {
//      printf("leaf node:%d\n", idx);
      for ( int i = 0; i < dtree[idx].trinum; i++ )
      {
       // printf("%d ", dtree[idx].tri[i]);

        idx0 =  dTriVtxBuf[3*dtree[idx].tri[i]];

        p0.x = dVtxBuf[3*idx0];
        p0.y = dVtxBuf[3*idx0+1];
        p0.z = dVtxBuf[3*idx0+2];

        idx0  = dTriVtxBuf[3*dtree[idx].tri[i]+1];
        p1.x = dVtxBuf[3*idx0];
        p1.y = dVtxBuf[3*idx0+1];
        p1.z = dVtxBuf[3*idx0+2];

        idx0 = dTriVtxBuf[3*dtree[idx].tri[i]+2];
        p2.x = dVtxBuf[3*idx0];
        p2.y = dVtxBuf[3*idx0+1];
        p2.z = dVtxBuf[3*idx0+2];

        t = TestSingleIntersection( p, dir ,p0, p1, p2); 
    
        if ( (t > 1e-5) && ( t < tmin ) )
        {
          intersectedTri = dtree[idx].tri[i];
          intersectedPt.x = p.x + t*dir.x;
          intersectedPt.y = p.y + t*dir.y;
          intersectedPt.z = p.z + t*dir.z;
          tmin = t;
        }
        
      }
//      printf("\n");
    }

    if ( intersectedTri >= 0 ) break;

    // second intersected point of the current leaf node
    // face: the intersected face of the current leaf ndoe
    IntersectBox(pt, dir, dtree, idx, pt, face, face);  

    // next node adjacent to the leaf
    idx = dtree[idx].rope[face];
    if ( (face==1) || (face==3) || (face==5) ) face --;
    else face++;
  }
}

__device__ inline uchar4 computeColor( int depth, float3 p, float3 dir, float* dVtxBuf, int* dTriVtxBuf, float* dNormal, float* dLS, KdTree_rp* dtree  )
{
  int tri = -1;
  float3 intersectPt;
  int triOnTheWay = -1;
  float3 intersectPtOnTheWay;

  float3 tempDir; 

  
  FindIntersectedTriangle(p,dir, tri, intersectPt, dVtxBuf, dTriVtxBuf, dtree); 

  uchar4 color;
  color.w= 0;
  color.x=0; color.y = 0; color.z= 0;

  int i =0;

  float l;
  l = sqrt( dir.x * dir.x + dir.y*dir.y + dir.z*dir.z);
  dir.x = dir.x /l;
  dir.y = dir.y / l;
  dir.z = dir.z /l;
    
  if ( tri < 0) // no intersection, return sandbox color
  {
    int face = -1;

    float3 pt;
    if ( fabs(dir.y)>0)
    {
      l = (-1.0 -p.y )/ dir.y;
      pt.x = p.x + l*dir.x;
      pt.y = -1.0;
      pt.z = p.z + l * dir.z;
      if ( (pt.x >=-1.0) && (pt.x<=1.0) && (pt.z>=-1.0) && (pt.z<=1.0) )
      {
        face = 2; // ymin 
      if ( depth == 0) 
        {
        tempDir.x = dir.x;
        tempDir.y = -dir.y;
        tempDir.z = dir.z;
        color = computeColor(depth+1,pt, tempDir, dVtxBuf, dTriVtxBuf, dNormal, dLS, dtree); 
        color.x = color.x * 0.8;
        color.y = color.y * 0.8;
        color.z = color.z * 0.8;
        }
      }
    }

    if ( fabs(dir.y)>0)
    {
      l = (1.0 -p.y )/ dir.y;
      pt.x = p.x + l*dir.x;
      pt.y = 1.0;
      pt.z = p.z + l * dir.z;
      if ( (pt.x >=-1.0) && (pt.x<=1.0) && (pt.z>=-1.0) && (pt.z<=1.0) )
        face = 3; // ymax
    }

    if ( fabs(dir.x)>0)
    {
      l = (-1.0 - p.x ) /dir.x;
      pt.x = -1.0;
      pt.y = p.y + l * dir.y;
      pt.z = p.z + l * dir.z;
      if ( (pt.y>=-1.0) && (pt.y<=1.0) && (pt.z>=-1.0) && (pt.z<=1.0))
        face = 0; // xmin
    }

    if ( fabs(dir.x)>0)
    {
      l = (1.0 - p.x ) /dir.x;
      pt.x = 1.0;
      pt.y = p.y + l * dir.y;
      pt.z = p.z + l * dir.z;
      if ( (pt.y>=-1.0) && (pt.y<=1.0) && (pt.z>=-1.0) && (pt.z<=1.0))
        face = 1; // xmax
    }

    if ( fabs(dir.z)>0)
    {
      l = (1.0 - p.z ) /dir.z;
      pt.x = p.x + l*dir.x;
      pt.y = p.y + l*dir.y;
      pt.z = 1.0;
      if ( (pt.x>=-1.0) && (pt.x<=1.0) && (pt.y>=-1.0) && (pt.y<=1.0) )
        face = 5; // zmax
    }

    switch (face)
    {
      case 2:
        break;
      case 3:
        color.x = 0;
        color.y = 0;
        color.z = 0;
        break;
      case 0:
        color.x = 10;
        color.y = 100;
        color.z = 10;
        break;
      case 5:
        color.x = 100;
        color.y = 10;
        color.z = 10;
        break;
      case 1:
        color.x = 10;
        color.y = 10;
        color.z =100;
        break;
      default:
        break;
    }

  }
  else  // intersected with a triangle
  {
    // check shadow or illumination
    triOnTheWay = -1;
    for ( i = 0; i < nLS; i++) // for every light source
    {

      tempDir.x = dLS[3*i] - intersectPt.x;
      tempDir.y = dLS[3*i+1] - intersectPt.y;
      tempDir.z = dLS[3*i+2] - intersectPt.z;
      
      l = sqrt(tempDir.x*tempDir.x + tempDir.y * tempDir.y + tempDir.z * tempDir.z);
      tempDir.x = tempDir.x / l;
      tempDir.y = tempDir.y / l;
      tempDir.z = tempDir.z / l;
      

      l = tempDir.x*dNormal[3*tri] + tempDir.y*dNormal[3*tri+1] + tempDir.z*dNormal[3*tri+2];
      
      if ( l < -1e-5) continue;

      FindIntersectedTriangle( intersectPt, tempDir, triOnTheWay, intersectPtOnTheWay, dVtxBuf, dTriVtxBuf, dtree);

      if ( triOnTheWay >= 0 ) // in a shadow of triangle "triOnTheWay"
      {
        /*
        color.w=0;
        color.x = 100;
        color.y = 0;
        color.z = 0;
        */
      }
      else  // directly illuminated by the current light source
      {
        color.w = 0;

        if ( i == 0) l = 0.3;
        else if ( i == 1) l =  0.1;

        color.x = color.x+ l* objectColor.x ;
        color.y = color.y+ l* objectColor.y ; 
        color.z = color.z+ l *objectColor.z ;
      }
    }

    l =  0.5*fabs( dir.x*dNormal[3*tri] + dir.y *dNormal[3*tri+1] + dir.z*dNormal[3*tri+2] ); 
    color.x = color.x+ l * objectColor.x;
    color.y = color.y + l* objectColor.y;
    color.z = color.z + l*objectColor.z;
   /* 
    color.x =  objectColor.x;
    color.y =  objectColor.y;
    color.z = objectColor.z;
    */
  } 
  

  return color;
}

__global__ void TracingKernel( uchar4* pos, float* dVtxBuf, int* dTriVtxBuf, float* dNormal, float* dLS, KdTree_rp* dtree ) 
{
  int pixelx = blockIdx.x*blocksize_x + threadIdx.x;
  int pixely = blockIdx.y*blocksize_y + threadIdx.y;

  float3 dir;
//  dir.x = dwindown.x + hx*pixelx - dcamPos.x;
//  dir.y = dwindown.y + hy*pixely - dcamPos.y;
  dir.x = dwinup.x - hx*pixelx - dcamPos.x;
  dir.y = dwinup.y- hy*pixely - dcamPos.y;
  dir.z = dwindown.z - dcamPos.z;
  
  pos[ pixely*dwinwidth + pixelx ] = computeColor(0, dcamPos, dir, dVtxBuf, dTriVtxBuf, dNormal, dLS, dtree);

  __syncthreads();
}

__global__ void TracingKernel_test( uchar4* pos, float* dVtxBuf, int* dTriVtxBuf, float* dNormal, float* dLS, KdTree_rp* dtree ) 
{
  int pixelx = blockIdx.x*blocksize_x + threadIdx.x;
  int pixely = blockIdx.y*blocksize_y + threadIdx.y;

  float3 dir;
//  dir.x = dwindown.x + hx*pixelx - dcamPos.x;
 // dir.y = dwindown.y + hy*pixely - dcamPos.y;
  dir.x =  -0.1;
  dir.y = -1;
  dir.z = 1;
  
  pos[ pixely*dwinwidth + pixelx ] = computeColor(0, dcamPos, dir, dVtxBuf, dTriVtxBuf, dNormal, dLS, dtree);

  __syncthreads();
}

// Be sure to launch after setting const. and tex. memory
extern "C" void launch_kernel( uchar4* pos, CScene& scene )
{
  dim3 dimBlock(8,8);

  dim3 dimGrid;
  dimGrid.x = scene.m_winwidth / dimBlock.x;
  dimGrid.y = scene.m_winheight / dimBlock.y;

  TracingKernel<<<dimGrid,dimBlock>>>(pos, scene.dVtxBuf, scene.dTriVtxBuf, scene.dNormal, scene.dLS, scene.dtree); 

  hipError_t error = hipGetLastError();
  if (error != hipSuccess )
  {
    printf("Cuda Error:%s\n", hipGetErrorString(error));
    exit(-1);
  }
}

extern "C" void launch_kernel_test( uchar4* pos, CScene& scene )
{
  dim3 dimBlock(blocksize_x,blocksize_y);
//  dim3 dimBlock(1);
  

  dim3 dimGrid;
  dimGrid.x = scene.m_winwidth / dimBlock.x;
  dimGrid.y = scene.m_winheight / dimBlock.y;

  uchar4* dpos;

  hipMalloc( (void**) &dpos, sizeof(uchar4)* scene.m_winwidth*scene.m_winheight );
  hipError_t error = hipGetLastError();
  if (error != hipSuccess )
  {
    printf("Cuda Error:%s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy( dpos, pos, sizeof(uchar4)*scene.m_winwidth*scene.m_winheight, hipMemcpyHostToDevice);
  error = hipGetLastError();
  if (error != hipSuccess )
  {
    printf("Cuda Error:%s\n", hipGetErrorString(error));
    exit(-1);
  }

  printf("Tracing...\n");
  float GPU_time = 0;
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start, 0 );

  TracingKernel<<<dimGrid,dimBlock>>>(dpos, scene.dVtxBuf, scene.dTriVtxBuf, scene.dNormal, scene.dLS, scene.dtree); 
 // TracingKernel_test<<<dimGrid,dimBlock>>>(dpos, scene.dVtxBuf, scene.dTriVtxBuf, scene.dNormal, scene.dLS, scene.dtree); 

  hipEventRecord( stop, 0);
  hipEventSynchronize( stop );
  hipEventElapsedTime( &GPU_time, start, stop);
  printf("GPU time:%f\n", GPU_time);

  printf("%d %d\n", scene.m_winwidth, scene.m_winheight);
  printf("DeviceToHost..\n");
  hipMemcpy( pos, dpos, sizeof(uchar4)*scene.m_winwidth*scene.m_winheight, hipMemcpyDeviceToHost);
  printf("DeviceToHostDone.\n");

  printf("Free...\n");
  hipFree(dpos);
  printf("Free done.\n");

  error = hipGetLastError();
  if (error != hipSuccess )
  {
    printf("Cuda Error:%s\n", hipGetErrorString(error));
    exit(-1);
  }
}

extern "C" void cudaSetConstantMem( CScene& scene )
{
  int a = scene.TriVtxBuf[0][0];  
  float3 camPos;
  camPos.x = scene.cameraPos.x;
  camPos.y = scene.cameraPos.y;
  camPos.z = scene.cameraPos.z;
  hipMemcpyToSymbol(HIP_SYMBOL(dcamPos), &camPos, sizeof(float3));

  float3 wincenterPos;
  wincenterPos.x = scene.windowCenter.x;
  wincenterPos.y = scene.windowCenter.y;
  wincenterPos.z = scene.windowCenter.z;
  hipMemcpyToSymbol(HIP_SYMBOL(dwincenterPos), &wincenterPos, sizeof(float3));

  float3 winup;
  winup.x = scene.window_diagup.x;
  winup.y = scene.window_diagup.y;
  winup.z = scene.window_diagup.z;
  hipMemcpyToSymbol(HIP_SYMBOL(dwinup), &winup, sizeof(float3)); 

  float3 windown;
  windown.x = scene.window_diagdown.x;
  windown.y = scene.window_diagdown.y;
  windown.z = scene.window_diagdown.z;
  hipMemcpyToSymbol(HIP_SYMBOL(dwindown), &windown, sizeof(float3));

  float3 boxup;
  boxup.x = scene.m_sandbox.m_diagup.x;
  boxup.y = scene.m_sandbox.m_diagup.y;
  boxup.z = scene.m_sandbox.m_diagup.z;
  hipMemcpyToSymbol(HIP_SYMBOL(dboxup), &boxup, sizeof(float3));

  float3 boxdown;
  boxdown.x = scene.m_sandbox.m_diagdown.x;
  boxdown.y = scene.m_sandbox.m_diagdown.y;
  boxdown.z = scene.m_sandbox.m_diagdown.z;
  hipMemcpyToSymbol(HIP_SYMBOL(dboxdown), &boxdown, sizeof(float3));

  float3 objColor;
  objColor.x = scene.objectColor.r;
  objColor.y = scene.objectColor.g;
  objColor.z = scene.objectColor.b;
  hipMemcpyToSymbol(HIP_SYMBOL(objectColor), &objColor, sizeof(float3));

  float hosthx = (winup.x-windown.x) / scene.m_winwidth;
  float hosthy = (winup.y - windown.y) / scene.m_winheight;
  hipMemcpyToSymbol(HIP_SYMBOL(hx), &hosthx, sizeof(hosthx));
  hipMemcpyToSymbol(HIP_SYMBOL(hy), &hosthy, sizeof(hosthy));

  hipMemcpyToSymbol(HIP_SYMBOL(dwinwidth), &scene.m_winwidth, sizeof(scene.m_winwidth));
  hipMemcpyToSymbol(HIP_SYMBOL(dwinheight), &scene.m_winheight, sizeof(scene.m_winheight));

  hipMemcpyToSymbol(HIP_SYMBOL(nTri), &scene.nTri, sizeof(unsigned int) );
  hipMemcpyToSymbol(HIP_SYMBOL(nVtx), &scene.nVtx, sizeof(&scene.nVtx) );
  hipMemcpyToSymbol(HIP_SYMBOL(nLS), &scene.nLightSource, sizeof(unsigned int) );

/*
  hipMemcpyToSymbol(HIP_SYMBOL( dVtxBuf), &scene.dVtxBuf, sizeof( scene.dVtxBuf ) ); 
  hipMemcpyToSymbol(HIP_SYMBOL( dTriVtxBuf), &scene.dTriVtxBuf, sizeof( scene.dTriVtxBuf ) );
  hipMemcpyToSymbol(HIP_SYMBOL( dNormal), &scene.dNormal, sizeof(scene.dNormal) );
  hipMemcpyToSymbol(HIP_SYMBOL( dLS), &scene.dLS, sizeof(scene.dLS) );
  hipMemcpyToSymbol(HIP_SYMBOL( dsandboxColor), &scene.dsandboxColor, sizeof(scene.dsandboxColor) );
  hipMemcpyToSymbol(HIP_SYMBOL( dsandboxIsReflective), &scene.dsandboxIsReflective, sizeof(scene.dsandboxIsReflective) );
  */
}

extern "C" void cudaSceneMalloc( CScene& scene )
{
  hipMalloc( (void**) & scene.dVtxBuf, sizeof(float)*scene.nVtx*3);
  hipMalloc( (void**) & scene.dTriVtxBuf, sizeof(int)*scene.nTri*3);
  hipMalloc((void**) &scene.dNormal, sizeof(float)*scene.nTri*3);
  hipMalloc((void**) &scene.dLS, sizeof(float)*3*scene.nLightSource);
  hipMalloc((void**) &scene.dsandboxColor, sizeof(float)*3*5);
  hipMalloc((void**) &scene.dsandboxIsReflective, sizeof(unsigned int)*5);
  hipMalloc((void**) &scene.dtree, sizeof(KdTree_rp)*scene.treesize);
}

extern "C" void cudaBindToTexture( unsigned int nVtx, unsigned int nTri, unsigned int nLS, CScene& scene )
{
  hipBindTexture(0, texref_VtxBuf, scene.dVtxBuf, sizeof(float)*nVtx*3);
  hipBindTexture(0, texref_TriVtx, scene.dTriVtxBuf, sizeof(int)*nTri*3);
  hipBindTexture(0, texref_Normal, scene.dNormal, sizeof(float)*3*nTri);
  hipBindTexture(0, texref_LS, scene.dLS, sizeof(float)*3*nLS); 
  hipBindTexture(0, texref_sandboxColor, scene.dsandboxColor, sizeof(float)*3*5);
  hipBindTexture(0, texref_sandboxIsReflective, scene.dsandboxColor, sizeof(unsigned int)*5);
}

extern "C" void cudaPassSceneToGlobalMem( CScene& scene,  float* pVtxBuf, int* pTriVtxBuf, float* pNormal, float* pLS, float* psandboxColor, unsigned int* psandboxIsReflective, KdTree_rp* ptree)
{
  if (scene.dVtxBuf)  hipMemcpy( scene.dVtxBuf, pVtxBuf, sizeof(float)*scene.nVtx*3, hipMemcpyHostToDevice);

  if (scene.dTriVtxBuf) hipMemcpy( scene.dTriVtxBuf, pTriVtxBuf, sizeof(int)*scene.nTri*3, hipMemcpyHostToDevice);

  if (scene.dNormal) hipMemcpy( scene.dNormal, pNormal, sizeof(float)*scene.nTri*3, hipMemcpyHostToDevice);

  if (scene.dLS) hipMemcpy( scene.dLS, pLS, sizeof(float)*3* scene.nLightSource, hipMemcpyHostToDevice);

  if (scene.dsandboxColor) hipMemcpy( scene.dsandboxColor, psandboxColor, sizeof(float3)*5, hipMemcpyHostToDevice );

  if (scene.dsandboxIsReflective)  hipMemcpy( scene.dsandboxIsReflective, psandboxIsReflective, sizeof(unsigned int)*5, hipMemcpyHostToDevice);

  if (scene.dtree) hipMemcpy( scene.dtree, ptree, sizeof(KdTree_rp)*scene.treesize, hipMemcpyHostToDevice );
}

// called when entire application ends
extern "C" void cudaFreeTextureResources()
{
  hipUnbindTexture(texref_VtxBuf);
  hipUnbindTexture(texref_Normal);
  hipUnbindTexture(texref_TriVtx);
  hipUnbindTexture(texref_LS);
}

extern "C" void cudaFreeGlobalMemory( CScene& scene)
{
  if ( scene.dVtxBuf ) hipFree(scene.dVtxBuf);
  if ( scene.dTriVtxBuf )  hipFree(scene.dTriVtxBuf);
  if ( scene.dNormal )  hipFree(scene.dNormal);
  if ( scene.dLS )  hipFree(scene.dLS);
  if ( scene.dsandboxColor ) hipFree(scene.dsandboxColor); 
  if ( scene.dsandboxIsReflective )  hipFree(scene.dsandboxIsReflective);
}

  
    


   
   
